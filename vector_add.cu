#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void cudaAddition(int* x, int* y, int* z, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        z[index] = x[index] + y[index];
    }
}

int main() {
    int N = 1e7;
    int *x = new int[N];
    int *y = new int[N];
    int *z = new int[N];

    for (int i = 0; i < N; i++) x[i] = 1, y[i] = 1;

    int *a, *b, *c;
    hipMalloc(&a, sizeof(int) * N);
    hipMalloc(&b, sizeof(int) * N);
    hipMalloc(&c, sizeof(int) * N);

    hipMemcpy(a, x, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(b, y, sizeof(int) * N, hipMemcpyHostToDevice);

    // Timing setup
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch kernel with timing
    dim3 th(256);                      // Use 256 threads per block
    dim3 bl((N + th.x - 1) / th.x);    // Compute number of blocks

    hipEventRecord(start);
    cudaAddition<<<bl, th>>>(a, b, c, N);
    hipEventRecord(stop);

    // Wait for kernel to finish
    hipEventSynchronize(stop);

    // Calculate time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time taken by CUDA kernel: " << milliseconds << " ms" << endl;

    hipMemcpy(z, c, sizeof(int) * N, hipMemcpyDeviceToHost);

    // Optional: Print results (you might want to print only first few for large N)
    // for (int i = 0; i < N; i++) cout << z[i] << " ";

    // Clean up
    delete[] x;
    delete[] y;
    delete[] z;
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
