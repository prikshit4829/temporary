
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void CudaAddition(int* x, int* y, int* z, int N){
      int index = blockIdx.x * blockDim.x + threadIdx.x;

      if (index < N) {
          z[index] = x[index]+y[index];
      }
}

int main(){
    int N = 1e6;

    // int *x = (int *)malloc(sizeof(int)*N);
    int *x = new int[N];
    int *y = new int[N];
    int *z = new int[N];
    // int *y = (int *)malloc(sizeof(int)*N);
    // int *z = (int *)malloc(sizeof(int)*N);
    cout<<"started"<<endl;

    for(int i=0;i<N;i++){
          x[i]=2;
          y[i]=2;
    }


    int *a , *b , *c;
    hipMalloc(&a , sizeof(int)*N);
    hipMalloc(&b , sizeof(int)*N);
    hipMalloc(&c , sizeof(int)*N);

    hipMemcpy(a, x, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(b, y, sizeof(int)*N, hipMemcpyHostToDevice);

    dim3 th(N);
    CudaAddition<<<1, th>>>(a, b, c, N);

    hipMemcpy(z, c, sizeof(int)*N, hipMemcpyDeviceToHost);

    // for(int i=0;i<N;i++){
    //     cout<<z[i]<<" ";
    // }

    cout<<"completed"<<endl;

    return 0;
}